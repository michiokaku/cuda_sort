#include "hip/hip_runtime.h"


#include <stdio.h>

__device__ __host__ unsigned int insert0_0(int a, int *array, int offset) {
	offset--;       offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_0(int a, int *array, int offset) {
	offset--;       offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_1(int a, int *array, int offset) {
	offset--;       offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_1(int a, int *array, int offset) {
	offset--;       offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_2(int a, int *array, int offset) {
	offset--;       offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_2(int a, int *array, int offset) {
	offset--;       offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_3(int a, int *array, int offset) {
	offset--;       offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_3(int a, int *array, int offset) {
	offset--;       offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_4(int a, int *array, int offset) {
	offset--;       offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_4(int a, int *array, int offset) {
	offset--;       offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_5(int a, int *array, int offset) {
	offset--;       offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_5(int a, int *array, int offset) {
	offset--;       offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_6(int a, int *array, int offset) {
	offset--;       offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_6(int a, int *array, int offset) {
	offset--;       offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_7(int a, int *array, int offset) {
	offset--;       offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_7(int a, int *array, int offset) {
	offset--;       offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_8(int a, int *array, int offset) {
	offset--;       offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_8(int a, int *array, int offset) {
	offset--;       offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_9(int a, int *array, int offset) {
	offset--;       offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_9(int a, int *array, int offset) {
	offset--;       offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_10(int a, int *array, int offset) {
	offset--;       offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_10(int a, int *array, int offset) {
	offset--;       offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_11(int a, int *array, int offset) {
	offset--;       offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_11(int a, int *array, int offset) {
	offset--;       offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_12(int a, int *array, int offset) {
	offset--;       offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_12(int a, int *array, int offset) {
	offset--;       offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_13(int a, int *array, int offset) {
	offset--;       offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_13(int a, int *array, int offset) {
	offset--;       offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_14(int a, int *array, int offset) {
	offset--;       offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_14(int a, int *array, int offset) {
	offset--;       offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_15(int a, int *array, int offset) {
	offset--;       offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_15(int a, int *array, int offset) {
	offset--;       offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_16(int a, int *array, int offset) {
	offset--;       offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_16(int a, int *array, int offset) {
	offset--;       offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_17(int a, int *array, int offset) {
	offset--;       offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_17(int a, int *array, int offset) {
	offset--;       offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_18(int a, int *array, int offset) {
	offset--;       offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_18(int a, int *array, int offset) {
	offset--;       offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_19(int a, int *array, int offset) {
	offset--;       offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_19(int a, int *array, int offset) {
	offset--;       offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_20(int a, int *array, int offset) {
	offset--;       offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_20(int a, int *array, int offset) {
	offset--;       offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_21(int a, int *array, int offset) {
	offset--;       offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_21(int a, int *array, int offset) {
	offset--;       offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_22(int a, int *array, int offset) {
	offset--;       offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_22(int a, int *array, int offset) {
	offset--;       offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_23(int a, int *array, int offset) {
	offset--;       offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_23(int a, int *array, int offset) {
	offset--;       offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_24(int a, int *array, int offset) {
	offset--;       offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_24(int a, int *array, int offset) {
	offset--;       offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_25(int a, int *array, int offset) {
	offset--;       offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_25(int a, int *array, int offset) {
	offset--;       offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_26(int a, int *array, int offset) {
	offset--;       offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_26(int a, int *array, int offset) {
	offset--;       offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_27(int a, int *array, int offset) {
	offset--;       offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_27(int a, int *array, int offset) {
	offset--;       offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_28(int a, int *array, int offset) {
	offset--;       offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_28(int a, int *array, int offset) {
	offset--;       offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_29(int a, int *array, int offset) {
	offset--;       offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_29(int a, int *array, int offset) {
	offset--;       offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_30(int a, int *array, int offset) {
	offset--;       offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_30(int a, int *array, int offset) {
	offset--;       offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_31(int a, int *array, int offset) {
	offset--;       offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_31(int a, int *array, int offset) {
	offset--;       offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_32(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_32(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_33(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_33(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_34(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_34(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_35(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_35(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_36(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_36(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_37(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_37(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_38(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_38(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_39(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_39(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_40(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_40(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_41(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_41(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_42(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_42(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_43(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_43(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_44(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_44(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_45(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_45(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_46(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_46(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_47(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_47(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_48(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_48(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_49(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_49(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_50(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_50(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_51(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_51(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_52(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_52(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_53(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_53(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_54(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_54(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_55(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_55(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_56(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 36028797018963968 * (a > array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_56(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 36028797018963968 * (a >= array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_57(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 72057594037927936 * (a > array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a > array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_57(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 72057594037927936 * (a >= array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a >= array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_58(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 144115188075855872 * (a > array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a > array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a > array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_58(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 144115188075855872 * (a >= array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a >= array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a >= array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_59(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 288230376151711744 * (a > array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a > array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a > array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a > array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_59(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 288230376151711744 * (a >= array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a >= array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a >= array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a >= array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_60(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 576460752303423488 * (a > array[offset + 576460752303423488]);
	offset += 288230376151711744 * (a > array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a > array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a > array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a > array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_60(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 576460752303423488 * (a >= array[offset + 576460752303423488]);
	offset += 288230376151711744 * (a >= array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a >= array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a >= array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a >= array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_61(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 1152921504606846976 * (a > array[offset + 1152921504606846976]);
	offset += 576460752303423488 * (a > array[offset + 576460752303423488]);
	offset += 288230376151711744 * (a > array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a > array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a > array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a > array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_61(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 1152921504606846976 * (a >= array[offset + 1152921504606846976]);
	offset += 576460752303423488 * (a >= array[offset + 576460752303423488]);
	offset += 288230376151711744 * (a >= array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a >= array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a >= array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a >= array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_62(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 2305843009213693952 * (a > array[offset + 2305843009213693952]);
	offset += 1152921504606846976 * (a > array[offset + 1152921504606846976]);
	offset += 576460752303423488 * (a > array[offset + 576460752303423488]);
	offset += 288230376151711744 * (a > array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a > array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a > array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a > array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_62(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 2305843009213693952 * (a >= array[offset + 2305843009213693952]);
	offset += 1152921504606846976 * (a >= array[offset + 1152921504606846976]);
	offset += 576460752303423488 * (a >= array[offset + 576460752303423488]);
	offset += 288230376151711744 * (a >= array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a >= array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a >= array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a >= array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert0_63(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 4611686018427387904 * (a > array[offset + 4611686018427387904]);
	offset += 2305843009213693952 * (a > array[offset + 2305843009213693952]);
	offset += 1152921504606846976 * (a > array[offset + 1152921504606846976]);
	offset += 576460752303423488 * (a > array[offset + 576460752303423488]);
	offset += 288230376151711744 * (a > array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a > array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a > array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a > array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a > array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a > array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a > array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a > array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a > array[offset + 1125899906842624]);
	offset += 562949953421312 * (a > array[offset + 562949953421312]);
	offset += 281474976710656 * (a > array[offset + 281474976710656]);
	offset += 140737488355328 * (a > array[offset + 140737488355328]);
	offset += 70368744177664 * (a > array[offset + 70368744177664]);
	offset += 35184372088832 * (a > array[offset + 35184372088832]);
	offset += 17592186044416 * (a > array[offset + 17592186044416]);
	offset += 8796093022208 * (a > array[offset + 8796093022208]);
	offset += 4398046511104 * (a > array[offset + 4398046511104]);
	offset += 2199023255552 * (a > array[offset + 2199023255552]);
	offset += 1099511627776 * (a > array[offset + 1099511627776]);
	offset += 549755813888 * (a > array[offset + 549755813888]);
	offset += 274877906944 * (a > array[offset + 274877906944]);
	offset += 137438953472 * (a > array[offset + 137438953472]);
	offset += 68719476736 * (a > array[offset + 68719476736]);
	offset += 34359738368 * (a > array[offset + 34359738368]);
	offset += 17179869184 * (a > array[offset + 17179869184]);
	offset += 8589934592 * (a > array[offset + 8589934592]);
	offset += 4294967296 * (a > array[offset + 4294967296]);
	offset += 2147483648 * (a > array[offset + 2147483648]);
	offset += 1073741824 * (a > array[offset + 1073741824]);
	offset += 536870912 * (a > array[offset + 536870912]);
	offset += 268435456 * (a > array[offset + 268435456]);
	offset += 134217728 * (a > array[offset + 134217728]);
	offset += 67108864 * (a > array[offset + 67108864]);
	offset += 33554432 * (a > array[offset + 33554432]);
	offset += 16777216 * (a > array[offset + 16777216]);
	offset += 8388608 * (a > array[offset + 8388608]);
	offset += 4194304 * (a > array[offset + 4194304]);
	offset += 2097152 * (a > array[offset + 2097152]);
	offset += 1048576 * (a > array[offset + 1048576]);
	offset += 524288 * (a > array[offset + 524288]);
	offset += 262144 * (a > array[offset + 262144]);
	offset += 131072 * (a > array[offset + 131072]);
	offset += 65536 * (a > array[offset + 65536]);
	offset += 32768 * (a > array[offset + 32768]);
	offset += 16384 * (a > array[offset + 16384]);
	offset += 8192 * (a > array[offset + 8192]);
	offset += 4096 * (a > array[offset + 4096]);
	offset += 2048 * (a > array[offset + 2048]);
	offset += 1024 * (a > array[offset + 1024]);
	offset += 512 * (a > array[offset + 512]);
	offset += 256 * (a > array[offset + 256]);
	offset += 128 * (a > array[offset + 128]);
	offset += 64 * (a > array[offset + 64]);
	offset += 32 * (a > array[offset + 32]);
	offset += 16 * (a > array[offset + 16]);
	offset += 8 * (a > array[offset + 8]);
	offset += 4 * (a > array[offset + 4]);
	offset += 2 * (a > array[offset + 2]);
	offset += 1 * (a > array[offset + 1]);
	offset++;
	return offset;
}


__device__ __host__ unsigned int insert1_63(int a, int *array, unsigned long long int offset) {
	offset--;       offset += 4611686018427387904 * (a >= array[offset + 4611686018427387904]);
	offset += 2305843009213693952 * (a >= array[offset + 2305843009213693952]);
	offset += 1152921504606846976 * (a >= array[offset + 1152921504606846976]);
	offset += 576460752303423488 * (a >= array[offset + 576460752303423488]);
	offset += 288230376151711744 * (a >= array[offset + 288230376151711744]);
	offset += 144115188075855872 * (a >= array[offset + 144115188075855872]);
	offset += 72057594037927936 * (a >= array[offset + 72057594037927936]);
	offset += 36028797018963968 * (a >= array[offset + 36028797018963968]);
	offset += 18014398509481984 * (a >= array[offset + 18014398509481984]);
	offset += 9007199254740992 * (a >= array[offset + 9007199254740992]);
	offset += 4503599627370496 * (a >= array[offset + 4503599627370496]);
	offset += 2251799813685248 * (a >= array[offset + 2251799813685248]);
	offset += 1125899906842624 * (a >= array[offset + 1125899906842624]);
	offset += 562949953421312 * (a >= array[offset + 562949953421312]);
	offset += 281474976710656 * (a >= array[offset + 281474976710656]);
	offset += 140737488355328 * (a >= array[offset + 140737488355328]);
	offset += 70368744177664 * (a >= array[offset + 70368744177664]);
	offset += 35184372088832 * (a >= array[offset + 35184372088832]);
	offset += 17592186044416 * (a >= array[offset + 17592186044416]);
	offset += 8796093022208 * (a >= array[offset + 8796093022208]);
	offset += 4398046511104 * (a >= array[offset + 4398046511104]);
	offset += 2199023255552 * (a >= array[offset + 2199023255552]);
	offset += 1099511627776 * (a >= array[offset + 1099511627776]);
	offset += 549755813888 * (a >= array[offset + 549755813888]);
	offset += 274877906944 * (a >= array[offset + 274877906944]);
	offset += 137438953472 * (a >= array[offset + 137438953472]);
	offset += 68719476736 * (a >= array[offset + 68719476736]);
	offset += 34359738368 * (a >= array[offset + 34359738368]);
	offset += 17179869184 * (a >= array[offset + 17179869184]);
	offset += 8589934592 * (a >= array[offset + 8589934592]);
	offset += 4294967296 * (a >= array[offset + 4294967296]);
	offset += 2147483648 * (a >= array[offset + 2147483648]);
	offset += 1073741824 * (a >= array[offset + 1073741824]);
	offset += 536870912 * (a >= array[offset + 536870912]);
	offset += 268435456 * (a >= array[offset + 268435456]);
	offset += 134217728 * (a >= array[offset + 134217728]);
	offset += 67108864 * (a >= array[offset + 67108864]);
	offset += 33554432 * (a >= array[offset + 33554432]);
	offset += 16777216 * (a >= array[offset + 16777216]);
	offset += 8388608 * (a >= array[offset + 8388608]);
	offset += 4194304 * (a >= array[offset + 4194304]);
	offset += 2097152 * (a >= array[offset + 2097152]);
	offset += 1048576 * (a >= array[offset + 1048576]);
	offset += 524288 * (a >= array[offset + 524288]);
	offset += 262144 * (a >= array[offset + 262144]);
	offset += 131072 * (a >= array[offset + 131072]);
	offset += 65536 * (a >= array[offset + 65536]);
	offset += 32768 * (a >= array[offset + 32768]);
	offset += 16384 * (a >= array[offset + 16384]);
	offset += 8192 * (a >= array[offset + 8192]);
	offset += 4096 * (a >= array[offset + 4096]);
	offset += 2048 * (a >= array[offset + 2048]);
	offset += 1024 * (a >= array[offset + 1024]);
	offset += 512 * (a >= array[offset + 512]);
	offset += 256 * (a >= array[offset + 256]);
	offset += 128 * (a >= array[offset + 128]);
	offset += 64 * (a >= array[offset + 64]);
	offset += 32 * (a >= array[offset + 32]);
	offset += 16 * (a >= array[offset + 16]);
	offset += 8 * (a >= array[offset + 8]);
	offset += 4 * (a >= array[offset + 4]);
	offset += 2 * (a >= array[offset + 2]);
	offset += 1 * (a >= array[offset + 1]);
	offset++;
	return offset;
}



#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include "insert.cuh"

#include <stdio.h>


#define blocksize 128

int getblock(int length) {
	int block = length / blocksize;
	if ((length%blocksize) > 0)block++;
	return block;
}

__device__ void merge_block(int *a_s, int thread_tid) {
	int r1 = 0, r2 = 0;//r1,r2�洢��ȡ����ֵ
	int p1 = 0, p2 = 0;//p1,p2Ϊ�����λ��
	int merge_offset = 0;//�ϲ����ƫ����
	int merge_tid = 0;//�߳��ںϲ����ж�Ӧ���̱߳��
	bool flag = (thread_tid < 256 / 2);
	thread_tid %= (256 / 2);//thread_tid��С��Ҫ����(blocksize/2)

	for (int i = 1; i < 256; i *= 2) {
		//�ȼ���merge_offset��merge_tid
		merge_offset = (thread_tid / i) * 4 * i;
		merge_tid = thread_tid%i;
		merge_tid += merge_offset;

		//��ȡr1,r2��ֵ
		if (flag) {
			r1 = a_s[merge_tid];
			r2 = a_s[merge_tid + 1];
		}
		else {
			r1 = a_s[merge_tid+i*2];
			r2 = a_s[merge_tid + 1+i*2];
		}

		__syncthreads();//��ȡ���һ��Ҫͬ��


		//�ҵ������λ��
	
		if (flag) {
			p1 = insert0_1(r1, a_s, merge_tid + 2*i)-i;
			p2 = insert0_1(r2, a_s, merge_tid + 2*i)-i+1;
		}
		else
		{
			p1 = insert1_1(r1, a_s, merge_tid);
			p2 = insert1_1(r2, a_s, merge_tid)+1;
		}

		//����p1,p2���в���
		a_s[p1] = r1;
		a_s[p2] = r2;
	}

}

__device__ void merge_block_256(int *a_s, int thread_tid) {
	int r1 = 0, r2 = 0;//r1,r2�洢��ȡ����ֵ
	int p1 = 0, p2 = 0;//p1,p2Ϊ�����λ��
	int merge_offset = 0;//�ϲ����ƫ����
	int merge_tid = 0;//�߳��ںϲ����ж�Ӧ���̱߳��
	bool flag = (thread_tid < 128);
	thread_tid %= 128;//thread_tid��С��Ҫ����(blocksize/2)
					  //�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 1) * 4;
	merge_tid = (thread_tid % 1) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 2];
		r2 = a_s[merge_tid + 3];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_1(r1, a_s, merge_tid + 2) - 1;

		p2 = insert0_1(r1, a_s, merge_tid + 2) - 0;

	}
	else
	{
		p1 = insert1_1(r1, a_s, merge_tid);
		p2 = insert1_1(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 2) * 8;
	merge_tid = (thread_tid % 2) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 4];
		r2 = a_s[merge_tid + 5];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_2(r1, a_s, merge_tid + 4) - 2;

		p2 = insert0_2(r1, a_s, merge_tid + 4) - 1;

	}
	else
	{
		p1 = insert1_2(r1, a_s, merge_tid);
		p2 = insert1_2(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 4) * 16;
	merge_tid = (thread_tid % 4) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 8];
		r2 = a_s[merge_tid + 9];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_3(r1, a_s, merge_tid + 8) - 4;

		p2 = insert0_3(r1, a_s, merge_tid + 8) - 3;

	}
	else
	{
		p1 = insert1_3(r1, a_s, merge_tid);
		p2 = insert1_3(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 8) * 32;
	merge_tid = (thread_tid % 8) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 16];
		r2 = a_s[merge_tid + 17];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_4(r1, a_s, merge_tid + 16) - 8;

		p2 = insert0_4(r1, a_s, merge_tid + 16) - 7;

	}
	else
	{
		p1 = insert1_4(r1, a_s, merge_tid);
		p2 = insert1_4(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 16) * 64;
	merge_tid = (thread_tid % 16) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 32];
		r2 = a_s[merge_tid + 33];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_5(r1, a_s, merge_tid + 32) - 16;

		p2 = insert0_5(r1, a_s, merge_tid + 32) - 15;

	}
	else
	{
		p1 = insert1_5(r1, a_s, merge_tid);
		p2 = insert1_5(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 32) * 128;
	merge_tid = (thread_tid % 32) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 64];
		r2 = a_s[merge_tid + 65];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_6(r1, a_s, merge_tid + 64) - 32;

		p2 = insert0_6(r1, a_s, merge_tid + 64) - 31;

	}
	else
	{
		p1 = insert1_6(r1, a_s, merge_tid);
		p2 = insert1_6(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 64) * 256;
	merge_tid = (thread_tid % 64) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 128];
		r2 = a_s[merge_tid + 129];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_7(r1, a_s, merge_tid + 128) - 64;

		p2 = insert0_7(r1, a_s, merge_tid + 128) - 63;

	}
	else
	{
		p1 = insert1_7(r1, a_s, merge_tid);
		p2 = insert1_7(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 128) * 512;
	merge_tid = (thread_tid % 128) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 256];
		r2 = a_s[merge_tid + 257];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_8(r1, a_s, merge_tid + 256) - 128;

		p2 = insert0_8(r1, a_s, merge_tid + 256) - 127;

	}
	else
	{
		p1 = insert1_8(r1, a_s, merge_tid);
		p2 = insert1_8(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

}

__device__ void merge_block_128(int *a_s, int thread_tid) {
	int r1 = 0, r2 = 0;//r1,r2�洢��ȡ����ֵ
	int p1 = 0, p2 = 0;//p1,p2Ϊ�����λ��
	int merge_offset = 0;//�ϲ����ƫ����
	int merge_tid = 0;//�߳��ںϲ����ж�Ӧ���̱߳��
	bool flag = (thread_tid < 64);
	thread_tid %= 64;//thread_tid��С��Ҫ����(blocksize/2)
					 //�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 1) * 4;
	merge_tid = (thread_tid % 1) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 2];
		r2 = a_s[merge_tid + 3];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_1(r1, a_s, merge_tid + 2) - 1;

		p2 = insert0_1(r1, a_s, merge_tid + 2) - 0;

	}
	else
	{
		p1 = insert1_1(r1, a_s, merge_tid);
		p2 = insert1_1(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 2) * 8;
	merge_tid = (thread_tid % 2) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 4];
		r2 = a_s[merge_tid + 5];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_2(r1, a_s, merge_tid + 4) - 2;

		p2 = insert0_2(r1, a_s, merge_tid + 4) - 1;

	}
	else
	{
		p1 = insert1_2(r1, a_s, merge_tid);
		p2 = insert1_2(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 4) * 16;
	merge_tid = (thread_tid % 4) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 8];
		r2 = a_s[merge_tid + 9];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_3(r1, a_s, merge_tid + 8) - 4;

		p2 = insert0_3(r1, a_s, merge_tid + 8) - 3;

	}
	else
	{
		p1 = insert1_3(r1, a_s, merge_tid);
		p2 = insert1_3(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 8) * 32;
	merge_tid = (thread_tid % 8) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 16];
		r2 = a_s[merge_tid + 17];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_4(r1, a_s, merge_tid + 16) - 8;

		p2 = insert0_4(r1, a_s, merge_tid + 16) - 7;

	}
	else
	{
		p1 = insert1_4(r1, a_s, merge_tid);
		p2 = insert1_4(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 16) * 64;
	merge_tid = (thread_tid % 16) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 32];
		r2 = a_s[merge_tid + 33];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_5(r1, a_s, merge_tid + 32) - 16;

		p2 = insert0_5(r1, a_s, merge_tid + 32) - 15;

	}
	else
	{
		p1 = insert1_5(r1, a_s, merge_tid);
		p2 = insert1_5(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 32) * 128;
	merge_tid = (thread_tid % 32) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 64];
		r2 = a_s[merge_tid + 65];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_6(r1, a_s, merge_tid + 64) - 32;

		p2 = insert0_6(r1, a_s, merge_tid + 64) - 31;

	}
	else
	{
		p1 = insert1_6(r1, a_s, merge_tid);
		p2 = insert1_6(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

	//�ȼ���merge_offset��merge_tid
	merge_offset = (thread_tid / 64) * 256;
	merge_tid = (thread_tid % 64) * 2;

	merge_tid += merge_offset;
	//��ȡr1,r2��ֵ
	if (flag) {
		r1 = a_s[merge_tid];
		r2 = a_s[merge_tid + 1];
	}
	else {
		r1 = a_s[merge_tid + 128];
		r2 = a_s[merge_tid + 129];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��

					//�ҵ������λ��
	if (flag) {
		p1 = insert0_7(r1, a_s, merge_tid + 128) - 64;

		p2 = insert0_7(r1, a_s, merge_tid + 128) - 63;

	}
	else
	{
		p1 = insert1_7(r1, a_s, merge_tid);
		p2 = insert1_7(r2, a_s, merge_tid) + 1;
	}
	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
	__syncthreads();



	////�µ�һ��

}



__global__ void sort_int_kernel_128(int *a_h, int *a_d, int length) {
	
	__shared__ int a_s[256];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int r1 = INT_MAX, r2 = INT_MAX;//��ʼֵ��Ϊ���ֵ��������������


	//��a_h��������ֵ
	tid *= 2;
	if (tid < length) {
		r1 = a_h[tid];
	}
	tid++;
	if (tid < length) {
		r2 = a_h[tid];
	}
	
	//��r1,r2����һ����򵥵�����
	if (r1 > r2) {
		int c = r1;
		r1 = r2;
		r2 = c;
	}

	//���빲���ڴ沢ͬ��
	int t2 = threadIdx.x * 2;
	a_s[t2] = r1;
	a_s[t2 + 1] = r2;
	__syncthreads();

	merge_block_128(a_s, threadIdx.x);
		

	__syncthreads();
	a_d[tid] = a_s[t2 + 1];
	a_d[tid - 1] = a_s[t2];
}

void sort_int(int *a,int lg2,int length) {
	int length_dev = 1 << lg2;
	printf("length_dev = %d\n", length_dev);
	int *a_dev;
	int *a_map;
	if (length_dev < blocksize)length_dev = blocksize;
	hipMalloc((void**)&a_dev, length_dev * sizeof(int));
	hipHostGetDevicePointer((void **)&a_map, (void *)a, 0);
	sort_int_kernel_128<<<getblock(length_dev/2),blocksize>>>(a_map, a_dev, length);
	hipMemcpy(a,a_dev,length*sizeof(int), hipMemcpyDeviceToHost);
}





int getlg2(int length) {
	int a = 0, b = 0;
	for (int i = 0; i < 32; i++) {
		if (((length >>i )&1) == 1) {
			printf("i = %d\n", i);
			a = i;
			b++;
		}
	}
	if (b > 1)a++;
	return a;
}

int* genarray(int length) {
	int *a;
	hipHostAlloc((void**)&a, length*sizeof(int), hipHostMallocMapped);
	for (int i = 0; i < length; i++) {
		a[i] =  i;
	};

	return a;
}



int main() {
	int length = 16;
	int *a = genarray(16);
	for (int i = 0; i < length; i++) {
		printf("a[%d] = %d\n", i, a[i]);
	}
	int lg2 = getlg2(length);
	printf("lg2 = %d\n", lg2);

	int in = insert0_4(33, a, 0);
	printf("in = %d \n", in);
	//sort_int(a,lg2,length);
	getchar();
}
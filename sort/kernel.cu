#include "hip/hip_runtime.h"




#include <stdio.h>


#define blocksize 256

int getblock(int length) {
	int block = length / blocksize;
	if ((length%blocksize) > 0)block++;
	return block;
}


int getlg2(int length);

void check(int *a, int length) {
	printf("//////////////////////////////////////////////\n");
	int t = 0;
	for (int i = 0; i < length; i++) {
		
		if (i < (length - 1)) {
			if (a[i] > a[i + 1])
				printf("a[%d] = %d , a[%d] = %d\n", i, a[i], i + 1, a[i + 1]);
				
		}
		else printf("succeed\n");
	}
}

__device__ __host__ int insert0(int a, int *array,unsigned int star,unsigned int end) {
	int p = 0;
	while ((end - star) > 1) {
		p = (star + end) / 2;
		if (a > array[p])star = p;
		else end = p;
	}
	p = (end + star)/2;
	if (a > array[p])star = p;
	else end = p;
	if (a > array[end])end++;
	return end;
}

__device__ __host__ int insert1(int a, int *array, unsigned int star, unsigned int end) {
	int p = 0;
	while ((end - star) > 1) {
		p = (star + end) / 2;
		if (a >= array[p])star = p;
		else end = p;
	}
	p = (end + star) / 2;
	if (a >= array[p])star = p;
	else end = p;
	if (a >= array[end])end++;
	return end;
}

__global__ void sort_int_shared(int *a_map, int *a_dev, int length) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int a_s[blocksize];

	//�Ȼ�ȡ�������ֵ���빲���ڴ���
	if(tid<length)a_s[threadIdx.x] = a_map[tid];
	else a_s[threadIdx.x] = INT_MAX;
	int r = 0;
	
	__syncthreads();

	//���߳̿�ֳ�����,�ֱ�ִ�в�ͬ������
	bool flag = (threadIdx.x < (blocksize / 2));
	tid = tid % (blocksize / 2);

	//����ֻ������Ԫ�ص����������
	if (flag) r = a_s[tid * 2];
	else r = a_s[tid * 2 + 1];
	__syncthreads();

	if (flag) {
		if(r > a_s[tid * 2 + 1])a_s[tid*2+1] = r;
	}
	else {
		if (r < a_s[tid * 2])a_s[tid * 2] = r;
	}
	__syncthreads();//�洢�������ͬ��

	for (int i = 2; i < blocksize; i *= 2) {
		int pair_star = (tid / i)*i*2;

		int offset = tid%i;
		r = a_s[pair_star + offset + ((!flag)*i)];

		int p;
		if (flag) {
			p = insert0(r, a_s, pair_star + i, pair_star + (2 * i) - 1);
			p = p - i;
		}
		else
		{
			p = insert1(r, a_s, pair_star, pair_star + i - 1);
		}

		__syncthreads();

		p += offset;
		a_s[p] = r;

		__syncthreads();

	}
	
	a_dev[blockIdx.x * blockDim.x + threadIdx.x] = a_s[threadIdx.x];
}

__global__ void short_int_global(int *a_dev, int *a_next, int i) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int r = a_dev[tid];
	int p = tid;

	bool flag = ((blockIdx.x >> i)&1);
	flag = !flag;
	int pair_length = 2 << i;
	int pair_tid = blockIdx.x / pair_length;
	if (flag) 
	{
		if ((blockIdx.x >> (i + 1)) == (gridDim.x >> (i + 1))) {
			int size = gridDim.x % (2 << i);
			size -= (pair_length / 2);
			if (size > 0) {
				int star = pair_tid*pair_length + (pair_length / 2);
				int end = star + size;
				star *= blocksize;
				end *= blocksize;
				end--;
				p = insert0(r, a_dev, star, end);
				p = p - blocksize*(pair_length / 2);
				p += tid - (pair_tid*pair_length*blocksize);
			}
			a_next[p] = r;
		}
		else
		{
			int star = pair_tid*pair_length + (pair_length / 2);
			int end = star + (pair_length/2);
			star *= blocksize;
			end *= blocksize;
			end--;
			p = insert0(r, a_dev, star, end);
			p = p - blocksize*(pair_length / 2);
			p += tid - (pair_tid*pair_length*blocksize);
			a_next[p] = r;
		}
	}
	else
	{
		int star = pair_tid*pair_length;
		int end = star + (pair_length / 2);
		star *= blocksize;
		end *= blocksize;
		end--;
		p = insert1(r, a_dev, star, end);
		p += tid - ((pair_tid*pair_length + (pair_length / 2))*blocksize);
		a_next[p] = r;
	}
}

void sort_int(int *a,int length) {;
	int *a_dev;
	int *a_map;
	hipMalloc((void**)&a_dev, getblock(length) * blocksize * sizeof(int));
	hipHostGetDevicePointer((void **)&a_map, (void *)a, 0);
	sort_int_shared<<<getblock(length),blocksize>>>(a_map, a_dev, length);
	
	hipMemcpy(a, a_dev, length * sizeof(int), hipMemcpyDeviceToHost);
	int *a_next;
	hipMalloc((void**)&a_next, getblock(length) * blocksize * sizeof(int));

	int times = getlg2(getblock(length));

	for (int i = 0; i < times; i++) {
		short_int_global <<<getblock(length), blocksize >>>(a_dev, a_next, i);
		//cudaMemcpy(a, a_next, length * sizeof(int), cudaMemcpyDeviceToHost);
		//check(a, length);

		if (i == (times - 1)) {
			hipMemcpy(a, a_next, length * sizeof(int), hipMemcpyDeviceToHost);
			break;
		}
		int *c;
		c = a_dev;
		a_dev = a_next;
		a_next = c;
	}

	hipFree(a_next);
	//cudaFree(a_dev);
	//cudaMemcpy(a,a_next,length*sizeof(int), cudaMemcpyDeviceToHost);
	hipFree(a_next);
	//check(a, length);
}





int getlg2(int length) {
	int a = 0, b = 0;
	for (int i = 0; i < 32; i++) {
		if (((length >>i )&1) == 1) {
			//printf("i = %d\n", i);
			a = i;
			b++;
		}
	}
	if (b > 1)a++;
	return a;
}

int* genarray(int length) {
	int *a;
	hipHostAlloc((void**)&a, length*sizeof(int), hipHostMallocMapped);
	for (int i = 0; i < length; i++) {
		a[i] =  length - i;
	};

	return a;
}



int main() {
	int length = 134217728;
	int *a = genarray(length);
	/*for (int i = 0; i < length; i++) {
		printf("a[%d] = %d\n", i, a[i]);
	}*/

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// record start event on the default stream
	hipEventRecord(start);

	sort_int(a,length);

	hipEventRecord(stop);
	// wait until the stop event completes
	hipEventSynchronize(stop);
	// calculate the elapsed time between two events
	float time;
	hipEventElapsedTime(&time, start, stop);

	printf("time = %f\n",time);

	getchar();
}
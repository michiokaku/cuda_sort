#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include "insert.cuh"

#include <stdio.h>


#define blocksize 256

int getblock(int length) {
	int block = length / blocksize;
	if ((length%blocksize) > 0)block++;
	return block;
}

__device__ void merge0_first(int* a_s, int tid) {//�����tid��ֵ���������ֵ
	int r1 = a_s[tid * 2];
	int r2 = a_s[tid * 2 + 1];

	int merge_pair = tid;//������һ��ϲ����ϲ������Ա��
	
	
}

__device__ void merge1_first(int* a_s, int tid) {
	int r1 = a_s[tid * 2+(blocksize/2)];
	int r2 = a_s[tid * 2 + 1+(blocksize/2)];

	int merge_pair = tid;
}

__global__ void sort_int_kernel(int *a_h, int *a_d, int length) {
	
	__shared__ int a_s[blocksize*2];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int r1 = INT_MAX, r2 = INT_MAX;//��ʼֵ��Ϊ���ֵ��������������


	//��a_h��������ֵ
	tid *= 2;
	if (tid < length) {
		r1 = a_h[tid];
	}
	tid++;
	if (tid < length) {
		r2 = a_h[tid];
	}
	
	//��r1,r2����һ����򵥵�����
	if (r1 > r2) {
		int c = r1;
		r1 = r2;
		r2 = c;
	}

	//���빲���ڴ沢ͬ��
	a_s[threadIdx.x] = r1;
	a_s[threadIdx.x + 1] = r2;
	__syncthreads();

	//ִ�в�ͬ�ĺ������߳̾����ֿ����������
	if (threadIdx.x >= blocksize/2) {
		merge1_first(a_s, threadIdx.x % (blocksize / 2));
	}
	else
	{
		merge0_first(a_s, threadIdx.x % (blocksize / 2));
	}

	
	a_d[tid] = a_s[threadIdx.x + 1];
	a_d[tid - 1] = a_s[threadIdx.x];
}

void sort_int(int *a,int lg2,int length) {
	int length_dev = 1 << lg2;
	printf("length_dev = %d\n", length_dev);
	int *a_dev;
	int *a_map;
	hipMalloc((void**)&a_dev, length_dev * sizeof(int));
	hipHostGetDevicePointer((void **)&a_map, (void *)a, 0);
	sort_int_kernel<<<getblock(length),length/2>>>(a_map, a_dev, length);
	hipMemcpy(a,a_dev,length*sizeof(int), hipMemcpyDeviceToHost);
}



int getlg2(int length) {
	int a = 0, b = 0;
	for (int i = 0; i < 32; i++) {
		if (((length >>i )&1) == 1) {
			printf("i = %d\n", i);
			a = i;
			b++;
		}
	}
	if (b > 1)a++;
	return a;
}

int* genarray(int length) {
	int *a;
	hipHostAlloc((void**)&a, length*sizeof(int), hipHostMallocMapped);
	for (int i = 0; i < length; i++) {
		a[i] =  i*3;
	};

	return a;
}



int main() {
	int length = 16;
	int *a = genarray(16);
	for (int i = 0; i < length; i++) {
		printf("a[%d] = %d\n", i, a[i]);
	}
	int lg2 = getlg2(length);
	printf("lg2 = %d\n", lg2);

	int in = insert0_4(10, a, 0);
	printf("in = %d\n", in);
	sort_int(a,lg2,length);
	getchar();
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include "insert.cuh"

#include <stdio.h>


#define blocksize 256

int getblock(int length) {
	int block = length / blocksize;
	if ((length%blocksize) > 0)block++;
	return block;
}

__device__ void merge_block(int *a_s, int thread_tid) {
	int r1 = 0, r2 = 0;
	
	//�Ȼ�ȡr1,r2��ֵ
	bool flag = (thread_tid < blocksize / 2);
	if (flag) {
		thread_tid *= 4;
		r1 = a_s[thread_tid];
		r2 = a_s[thread_tid+1];
	}
	else
	{
		thread_tid %= (blocksize / 2);//thread_tid��С��Ҫ����(blocksize/2)
		thread_tid *= 4;
		r1 = a_s[thread_tid+2];
		r2 = a_s[thread_tid + 3];
	}
	__syncthreads();//��ȡ���һ��Ҫͬ��


	//�ҵ������λ��
	int p1 = 0, p2 = 0;//p1,p2Ϊ�����λ��
	if (flag) {
		p1 = insert0_1(r1, a_s, thread_tid + 2);
		p2 = insert0_1(r2, a_s, thread_tid + 2)+1;
	}
	else
	{
		p1 = insert1_1(r1, a_s, thread_tid);
		p2 = insert1_1(r2, a_s, thread_tid)+1;
	}

	//����p1,p2���в���
	a_s[p1] = r1;
	a_s[p2] = r2;
}


__global__ void sort_int_kernel(int *a_h, int *a_d, int length) {
	
	__shared__ int a_s[blocksize*2];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int r1 = INT_MAX, r2 = INT_MAX;//��ʼֵ��Ϊ���ֵ��������������


	//��a_h��������ֵ
	tid *= 2;
	if (tid < length) {
		r1 = a_h[tid];
	}
	tid++;
	if (tid < length) {
		r2 = a_h[tid];
	}
	
	//��r1,r2����һ����򵥵�����
	if (r1 > r2) {
		int c = r1;
		r1 = r2;
		r2 = c;
	}

	//���빲���ڴ沢ͬ��
	int t2 = threadIdx.x * 2;
	a_s[t2] = r1;
	a_s[t2 + 1] = r2;
	__syncthreads();

	merge_block(a_s, threadIdx.x);
		
	a_d[tid] = a_s[t2 + 1];
	a_d[tid - 1] = a_s[t2];
}

void sort_int(int *a,int lg2,int length) {
	int length_dev = 1 << lg2;
	printf("length_dev = %d\n", length_dev);
	int *a_dev;
	int *a_map;
	if (length_dev < blocksize)length_dev = blocksize;
	hipMalloc((void**)&a_dev, length_dev * sizeof(int));
	hipHostGetDevicePointer((void **)&a_map, (void *)a, 0);
	sort_int_kernel<<<getblock(length),blocksize>>>(a_map, a_dev, length);
	hipMemcpy(a,a_dev,length*sizeof(int), hipMemcpyDeviceToHost);
}



int getlg2(int length) {
	int a = 0, b = 0;
	for (int i = 0; i < 32; i++) {
		if (((length >>i )&1) == 1) {
			printf("i = %d\n", i);
			a = i;
			b++;
		}
	}
	if (b > 1)a++;
	return a;
}

int* genarray(int length) {
	int *a;
	hipHostAlloc((void**)&a, length*sizeof(int), hipHostMallocMapped);
	for (int i = 0; i < length; i++) {
		a[i] =  i;
	};

	return a;
}



int main() {
	int length = 16;
	int *a = genarray(16);
	for (int i = 0; i < length; i++) {
		printf("a[%d] = %d\n", i, a[i]);
	}
	int lg2 = getlg2(length);
	printf("lg2 = %d\n", lg2);

	int in = insert0_4(-1, a, 0);
	printf("in = %d \n", in);
	//sort_int(a,lg2,length);
	getchar();
}